#include "hip/hip_runtime.h"
#include "circuit_executer.h"
#include "gates.h"
#include "quantum_state.h"
#include "circuit.h"
#include <cmath>
#include <stdexcept>
#include <iostream>
#include <unordered_map>

const int THREADS_PER_BLOCK = 256; 

__global__ void applyGate(hipDoubleComplex* stateVec,const Gate* gates, int numQubits, int numGates); 

std::pair<dim3, dim3> CircuitExecuter::calculateLaunchDims(int total_elements) const {
    int num_blocks = (total_elements + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    dim3 grid(num_blocks);
    dim3 block(THREADS_PER_BLOCK);
    return {grid, block};
}


void CircuitExecuter::execute(Circuit& circuit, QuantumState& state) {
    int num_blocks = (state.num_amplitudes + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    
    // allocate a hipDoubleComplex the size 4 * numGates * sizeof(hipDoubleComplex)
    Gate* d_gates;
    size_t matSize = circuit.gates.size() * sizeof(Gate); 
    hipMalloc(&d_gates, matSize);
    hipMemcpy(d_gates, circuit.gates.data(), matSize, hipMemcpyHostToDevice);

    applyGate<<<num_blocks, THREADS_PER_BLOCK>>>(state.amplitudes, d_gates, circuit.qubit_count, static_cast<int>(circuit.gates.size()));

    hipFree(d_gates);

    hipDeviceSynchronize(); 
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA error after kernel launch: " << hipGetErrorString(err) << std::endl;
    }
}

__global__ void applyGate(hipDoubleComplex* stateVec,const Gate* gates, int numQubits, int numGates) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int dim = 1 << numQubits;  
    if (idx >= dim) return;

    hipDoubleComplex amp = stateVec[idx];  

    for (int gate = 0; gate < numGates; gate++){
        int i = gates[gate].targets[0]; 
        int pairIdx = idx ^ (1 << i); 

        if(idx < pairIdx){
            hipDoubleComplex a = stateVec[idx]; 
            hipDoubleComplex b = stateVec[pairIdx]; 

            stateVec[idx] =     hipCadd(hipCmul(gates[gate].matrix[0], a), 
                                       hipCmul(gates[gate].matrix[1], b)); 
            
            stateVec[pairIdx] = hipCadd(hipCmul(gates[gate].matrix[2], a), 
                                       hipCmul(gates[gate].matrix[3], b)); 
        }
        __syncthreads(); 
    }
}
