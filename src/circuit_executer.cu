#include "hip/hip_runtime.h"
#include "circuit_executer.h"
#include "gates.h"
#include "quantum_state.h"
#include "circuit.h"
#include <cmath>
#include <stdexcept>
#include <iostream>
#include <unordered_map>

const int THREADS_PER_BLOCK = 256; 

__global__ void applyGate(hipDoubleComplex* stateVec, const hipDoubleComplex* U, int numQubits); 

dim3 CircuitExecuter::calculateLaunchDims(int total_elements) const {
    int num_blocks = static_cast<int>(std::ceil(static_cast<double>(total_elements) / THREADS_PER_BLOCK)); 
    return dim3(num_blocks, THREADS_PER_BLOCK); 
}

void CircuitExecuter::execute(Circuit& circuit, QuantumState& state) {
    dim3 launch_dims = calculateLaunchDims(state.num_amplitudes); 
    unsigned int num_blocks = launch_dims.x; 
    unsigned int num_threads = launch_dims.y; 
    
    prepareMatrix(circuit); 

    hipDoubleComplex* d_matrix;
    size_t matSize = circuit.completeMatrix.size() * sizeof(hipDoubleComplex);
    hipMalloc(&d_matrix, matSize);
    hipMemcpy(d_matrix, circuit.completeMatrix.data(), matSize, hipMemcpyHostToDevice);

    applyGate<<<num_blocks, num_threads>>>(state.amplitudes, d_matrix, circuit.qubit_count);

    hipFree(d_matrix);

    hipDeviceSynchronize(); 
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA error after kernel launch: " << hipGetErrorString(err) << std::endl;
    }
}

// Only handles single-qubit gates
void CircuitExecuter::prepareMatrix(Circuit& circuit) {
    int numQubits = circuit.qubit_count;

    std::vector<std::vector<hipDoubleComplex>> qubitMatrices(numQubits, std::vector<hipDoubleComplex>{
        {1,0}, {0,0},
        {0,0}, {1,0}
    });
    
    for (const Gate& gate : circuit.gates) {
        if (gate.targets.size() != 1) {
            throw std::runtime_error("Only single-qubit gates supported in prepareMatrix.");
        }
        int q = gate.targets[0];
        qubitMatrices[q] = multiplyMatrices(gate.matrix, qubitMatrices[q], 2, 2);
    }

    std::vector<hipDoubleComplex> finalMatrix = qubitMatrices[0];
    for (int q = 1; q < numQubits; ++q) {
        finalMatrix = tensorProduct(finalMatrix, qubitMatrices[q], 1 << q, 2);
    }

    circuit.completeMatrix = finalMatrix;
}

bool overlaps(const Gate& a, const Gate& b) {
    for (int qa: a.targets) {
        for(int qb: b.targets) {
            if (qa == qb) return true;
        }
    }
    return false;
}

std::vector<hipDoubleComplex> tensorProduct(const std::vector<hipDoubleComplex>& A, const std::vector<hipDoubleComplex>& B, int dimA, int dimB) {
    std::vector<hipDoubleComplex> result(dimA * dimB * dimA * dimB);
    for (int i = 0; i < dimA; ++i) {
        for (int j = 0; j < dimA; ++j) {
            for (int k = 0; k < dimB; ++k) {
                for (int l = 0; l < dimB; ++l) {
                    result[(i*dimB + k) * (dimA*dimB) + (j*dimB + l)] = hipCmul(A[i*dimA + j], B[k*dimB + l]);
                }
            }
        }
    }
    return result;
}

std::vector<hipDoubleComplex> multiplyMatrices(const hipDoubleComplex* A, const std::vector<hipDoubleComplex>& B, int dimA, int dimB) {
    std::vector<hipDoubleComplex> result(dimA*dimA, make_hipDoubleComplex(0,0));
    for (int i = 0; i < dimA; ++i) {
        for (int j = 0; j < dimA; ++j) {
            hipDoubleComplex sum = make_hipDoubleComplex(0,0);
            for (int k = 0; k < dimA; ++k) {
                sum = hipCadd(sum, hipCmul(A[i*dimA + k], B[k*dimA + j]));
            }
            result[i*dimA + j] = sum;
        }
    }
    return result;
}

__global__ void applyGate(hipDoubleComplex* stateVec, const hipDoubleComplex* U, int numQubits) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int dim = 1 << numQubits;
    if (idx >= dim) return;

    extern __shared__ hipDoubleComplex temp[];
    int tIdx = threadIdx.x;
    temp[tIdx] = stateVec[idx];
    __syncthreads();


    hipDoubleComplex sum = make_hipDoubleComplex(0,0);
    int gateDim = 1 << numQubits;
    for(int j = 0; j < gateDim; ++j) {
        sum = hipCadd(sum, hipCmul(U[idx * gateDim + j], temp[j]));
    }

    stateVec[idx] = sum;
}
