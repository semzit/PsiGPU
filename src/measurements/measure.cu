
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>

// Go through every amplitude and calculate probability
__global__ void calculate_probability(hipDoubleComplex* state_vector, double* probabilities, int num_qubits){
    int i = blockIdx.x*blockDim.x + threadIdx.x;

}

// go through every amplitude and set it to 0 instead of the greates one which is set to 1.0
__global__ void measure(hipDoubleComplex* state_vector, double* probabilities, int num_qubits){
    int i = blockIdx.x*blockDim.x + threadIdx.x;

}
